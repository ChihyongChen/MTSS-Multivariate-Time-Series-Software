#include "hip/hip_runtime.h"
#include "dtw_h.h"

using namespace std;

int main( int argc, char** argv)
{
	int i,nss; 
	hipEvent_t start,stop;
	float timeCPU,timeGM_GPU,timeSM_GPU;

    long int window_size;
    float t_size,q_size,n_attr_t,n_attr_q;

    int blockSize=atoi(argv[1]);

    if(argc<4){
        printf("Too few arguments!!!\n");
        exit(-1);
    }

    FILE *tFile,*qFile;
    tFile=fopen(argv[2],"r");
    if(tFile==NULL) printf("The file %s cannot be opened!\n", argv[2]);
    qFile=fopen(argv[3],"r");
    if(qFile==NULL) printf("The file %s cannot be opened!\n", argv[3]);


	//reading T and Q lengths series, and number of their attributes
    fscanf(tFile,"%f",&t_size);fscanf(tFile,"%f",&n_attr_t);
    fscanf(qFile,"%f",&q_size);fscanf(qFile,"%f",&n_attr_q);

    //nss=number of subsequences
    nss=t_size-q_size+1;

	window_size=q_size; //query size

	//T and Q lengths series bytes
	unsigned long long int t_bytes=t_size*sizeof(float);
	unsigned long long int q_bytes=q_size*sizeof(float);
	
	////////////////////////////////CPU MEMORY ALLOCATION////////////////////////////////
    float* x_t_serie = (float*) malloc (t_bytes);
	float* t_serie = (float*) malloc (t_bytes);
    float* x_q_serie = (float*) malloc (q_bytes);    
	float* q_serie = (float*) malloc (q_bytes);
	float* q_serie_norm = (float*) malloc (q_bytes);
	float* subseq_norm = (float*) malloc (q_bytes);
    float* owp = (float*) malloc (nss*sizeof(float));
    memset(owp,0,nss*sizeof(float));

    //Setting CUDA variables and structure
	float grid_size=ceil((double)nss/blockSize); 
	dim3 grid(grid_size,1); //number of blocks
	dim3 threads(blockSize,1); //number of threads for blocks


    infoDev(1);
    printf("\n****************Subsequence Search Parameters****************\n\n");
	printf("Path Time Series T: %s, length: %f, n_attrs: %f byte_size: %f\n",argv[2],t_size,n_attr_t,sizeof(float)*t_size);
	printf("Path Time Series Q: %s, length: %f, n_attrs: %f, byte_size: %f\n",argv[3],q_size,n_attr_q,sizeof(float)*q_size);
    printf("Number of Subsequences to search: %d\n", nss);
    printf("Windows size: %d\n", window_size);
    printf("Grid_size_x: %d, number_of_threads_x: %d \n", grid.x,threads.x);
    printf("Grid_size_y: %d, number_of_threads_y: %d \n\n", grid.y,threads.y);


    //T series file reading
    printf("Reading Time Series T...\t");
    for(i = 0; i < t_size; i++) {
        fscanf(tFile,"%f",&x_t_serie[i]);
        fscanf(tFile,"%f",&t_serie[i]);
    }
    fclose(tFile);
    printf("done!\n");

    printf("Reading Time Series Q...\t");
    //Q series file reading
    for(i = 0; i < q_size; i++) {
        fscanf(qFile,"%f",&x_q_serie[i]);
        fscanf(qFile,"%f",&q_serie[i]);
    }
    fclose(qFile);
    printf("done!\n");

    printf("Data has been read!\n\n");

    //Query normalization
    z_normalize(q_serie,q_size,q_serie_norm);


	////////////////////////////////DTW CPU ALGORITHM////////////////////////////////
    printf("DTW CPU version processing...\n");
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    for (int i = 0; i <nss;i++) {

    	float min=0.0;
    	z_normalize(&t_serie[i],window_size,subseq_norm);
    	min=short_dtw_c(subseq_norm,q_serie_norm,window_size,window_size);
    	owp[i]=min;
    }

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeCPU,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //computing minimum value
    int* ind_min_val = (int*) malloc (sizeof(int));
    float min_val = min_arr(owp,nss,ind_min_val);
    printf("ind_min_val_CPU_version: %d, min_val_CPU_version: %f\n\n",*ind_min_val,min_val);

    //owp's indices
    float* owp_ind = (float*) malloc (nss*sizeof(float));
    initializeArray(owp_ind,nss);


	////////////////////////////////DTW GPU_GM ALGORITHM////////////////////////////////
    float* d_t_serie = 0,*d_owp=0,* d_query_norm=0;
    hipMalloc((void**)&d_t_serie, t_bytes);
    hipMemcpy(d_t_serie, t_serie, t_bytes, hipMemcpyHostToDevice);
    hipMalloc((void**)&d_query_norm,q_bytes);
	hipMemcpy(d_query_norm, q_serie_norm, q_bytes, hipMemcpyHostToDevice); //already nornalized
	hipMalloc((void**)&d_owp,nss*sizeof(float));
	hipMemset(d_owp, 0, nss*sizeof(float));


    printf("DTW GPU_GM version processing...\n");
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    DTW_GM <<<grid, threads>>> (d_t_serie,d_query_norm,t_size,d_owp);
    checkCUDAError("Kernel DTW_GM");

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeGM_GPU,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    //TESTING
    float *d_owp_copy=(float*) malloc(nss*sizeof(float));
    hipMemcpy(d_owp_copy, d_owp,nss*sizeof(float),hipMemcpyDeviceToHost);
    min_val = min_arr(d_owp_copy,nss,ind_min_val);
    printf("ind_min_val_GPU_GM_version: %d, min_val_GPU_GM_version: %f\n\n",*ind_min_val,min_val);



    ////////////////////////DTW GPU_SM ALGORITHM//////////////////////// 
    float query_sm=(window_size)*sizeof(float);

    printf("DTW GPU_SM version processing...\n");
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start,0);

    DTW_SM <<<grid, threads, query_sm>>> (d_t_serie,d_query_norm,t_size,d_owp);
    checkCUDAError("Kernel DTW_SM1");

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&timeSM_GPU,start,stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    min_val = min_arr(d_owp_copy,nss,ind_min_val);
    printf("ind_min_val_GPU_SM_version: %d, min_val_GPU_SM_version: %f\n\n",*ind_min_val,min_val);
    

    //Execution time
    printf("Execution time for DTW with CPU %f ms\n",timeCPU);
    printf("Execution time for DTW with GPU_GM %f ms\n",timeGM_GPU);
    printf("Execution time for DTW with GPU_SM %f ms\n",timeSM_GPU);


    hipFree(d_t_serie);
    hipFree(d_query_norm);
    hipFree(d_t_serie);
    hipFree(d_owp);  

    free(t_serie);
    free(x_t_serie);    
    free(q_serie);
    free(x_q_serie);
    free(q_serie_norm);
    free(owp);
    free(owp_ind);
    printf("\nMemory deallocated!\n\n");

    return 0;
}	
